// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <gtest/gtest.h>
#include <algorithm>
#include <chrono>
#include <vector>

#include "paddle/fluid/framework/fleet/heter_ps/feature_value.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_ps_table.h"
#include "paddle/fluid/framework/fleet/heter_ps/graph_gpu_wrapper.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_comm.h"
#include "paddle/fluid/framework/fleet/heter_ps/heter_resource.h"
#include "paddle/fluid/framework/fleet/heter_ps/optimizer.cuh.h"
#include "paddle/fluid/platform/cuda_device_guard.h"

using namespace paddle::framework;
namespace platform = paddle::platform;
struct Edge {
  uint64_t from, to;
};
struct EdgeLess {
  bool operator()(const Edge &a, const Edge &b) {
    return a.from < b.from || a.from == b.from && a.to < b.to;
  }
};
void put_edges_to_file(std::string name, std::vector<Edge> &e) {
  // printf("%s\n", name.c_str());
  // for (int i = 0; i < e.size(); i++) {
  //   printf("%d %d\n", e[i].from, e[i].to);
  // }
  std::ofstream ofile;
  std::string file_name = "./" + name + ".txt";
  ofile.open(file_name.c_str());
  for (auto edge : e) {
    ofile << edge.from << "\t" << edge.to << std::endl;
  }
  ofile.close();
}
void generate_random_edge_input(std::vector<std::string> &edge_type,
                                std::vector<std::string> &node_type,
                                int node_each,
                                int edge_num,
                                std::map<Edge, int, EdgeLess> &edge_map,
                                std::vector<std::vector<Edge>> &edge_list) {
  srand(time(0));
  int s = 0;
  std::vector<int> from_type, to_type;
  std::map<std::string, int> node_type_map;
  for (int i = 0; i < node_type.size(); i++) {
    node_type_map[node_type[i]] = i;
  }
  for (auto str : edge_type) {
    auto pos = str.find("2");
    auto from_part = str.substr(0, pos);
    auto to_part = str.substr(pos + 1, str.size() - pos - 1);
    from_type.push_back(node_type_map[from_part]);
    to_type.push_back(node_type_map[to_part]);
  }
  int node_type_size = node_type_map.size();
  int edge_type_size = edge_type.size();
  int total_num = node_each * node_type_size;
  if (node_type_size == 0) return;
  edge_list.resize(edge_type_size);
  int empty_set_num = edge_type_size;
  int type_index;
  while (edge_num > 0 || empty_set_num != 0) {
    Edge temp;

    type_index = rand() % edge_type_size;
    temp.from = from_type[type_index] * node_each + rand() % node_each;
    temp.to = to_type[type_index] * node_each + rand() % node_each;

    if (edge_map.find(temp) != edge_map.end()) continue;
    edge_map[temp] = type_index;
    edge_list[type_index].push_back(temp);
    if (edge_list[type_index].size() == 1) empty_set_num--;
    edge_num--;
  }
  for (int i = 0; i < edge_type_size; i++) {
    put_edges_to_file(edge_type[i], edge_list[i]);
  }
}

TEST(TEST_FLEET, test_heter_graph) {
  auto iter = paddle::framework::GraphGpuWrapper::GetInstance();
  std::vector<int> device;
  int device_num = 2;
  for (int i = 0; i < device_num; i++) device.push_back(i);
  iter->set_device(device);
  const int edge_type_num = 9;
  const int node_type_num = 4;
  std::string edge_type_strs[edge_type_num] = {
      "a2b", "a2c", "b2c", "b2b", "c2a", "c2c", "d2a", "d2c", "d2d"};
  std::string node_type_strs[node_type_num] = {"a", "b", "c", "d"};
  std::vector<std::string> edge_types(edge_type_strs,
                                      edge_type_strs + edge_type_num);
  std::vector<std::string> node_types(node_type_strs,
                                      node_type_strs + node_type_num);
  int node_each = 50000;
  int sample_size = 5;
  int edge_num = (int64_t)node_each * node_each * edge_types.size() * 0.001;
  std::map<Edge, int, EdgeLess> edge_map;
  std::vector<std::vector<Edge>> edge_list;
  generate_random_edge_input(
      edge_types, node_types, node_each, edge_num, edge_map, edge_list);
  std::cerr << "begin to set up types" << std::endl;
  iter->set_up_types(edge_types, node_types);
  std::cerr << "init_service" << std::endl;
  iter->init_service();
  std::cerr << "init_service_over" << std::endl;
  for (int i = 0; i < edge_types.size(); i++)
    iter->load_edge_file(edge_types[i], "./" + edge_types[i] + ".txt", false);
  for (int i = 0; i < edge_types.size(); i++) {
    VLOG(0) << "upload edge_type " << edge_types[i];
    iter->upload_batch(0, i, device_num, edge_types[i]);
    VLOG(0) << "upload edge_type done " << edge_types[i];
  }
  auto edge_type_graph_ = iter->get_edge_type_graph(0, edge_types.size());
  srand(time(0));

  std::vector<uint64_t> key_vec;
  std::vector<int> node_type_vec;
  int total_range = node_types.size() * node_each;
  for (int i = 0; i < node_types.size() * node_each; i++) {
    key_vec.push_back(i);
    node_type_vec.push_back(i / node_each);
  }

  // if (key_vec.size() == 0) {
  //   key_vec.push_back(0);
  //   node_type_vec.push_back(0);
  // }
  for (int i = 0; i < total_range; i++) {
    int r = rand() % (total_range - i);
    std::swap(key_vec[i], key_vec[i + r]);
    std::swap(node_type_vec[i], node_type_vec[i + r]);
  }
  std::vector<int> edges_split_num;
  int edges_len;
  uint64_t *key;
  hipMalloc((void **)&key, sizeof(uint64_t) * key_vec.size());
  hipMemcpy(key,
             key_vec.data(),
             sizeof(uint64_t) * key_vec.size(),
             hipMemcpyHostToDevice);
  int *d_node_types;
  hipMalloc((void **)&d_node_types, sizeof(int) * node_type_vec.size());
  hipMemcpy(d_node_types,
             node_type_vec.data(),
             sizeof(int) * node_type_vec.size(),
             hipMemcpyHostToDevice);

  VLOG(0) << "begin to sample";
  auto res = iter->sample_neighbor_with_node_type(0,
                                                  key,
                                                  sample_size,
                                                  key_vec.size(),
                                                  edge_type_graph_,
                                                  d_node_types,
                                                  node_types.size(),
                                                  edges_len,
                                                  edges_split_num);
  // ASSERT_EQ(res.size(),0);
  int64_t *d_neighbors_ptr = reinterpret_cast<int64_t *>(res[0]->ptr());
  int64_t *d_index_ptr = reinterpret_cast<int64_t *>(res[1]->ptr());
  int *d_type_ptr = reinterpret_cast<int *>(res[2]->ptr());
  std::vector<uint64_t> h_neighbors, h_index;
  std::vector<int> h_node_type;
  h_neighbors.resize(edges_len);
  h_index.resize(edges_len);
  h_node_type.resize(edges_len);
  hipMemcpy(h_neighbors.data(),
             d_neighbors_ptr,
             edges_len * sizeof(int64_t),
             hipMemcpyDeviceToHost);
  hipMemcpy(h_index.data(),
             d_index_ptr,
             edges_len * sizeof(int64_t),
             hipMemcpyDeviceToHost);
  hipMemcpy(h_node_type.data(),
             d_type_ptr,
             edges_len * sizeof(int),
             hipMemcpyDeviceToHost);
  std::set<Edge, EdgeLess> query_edge_set;
  int last_edge_type = -1;
  for (int i = 0; i < edges_len; i++) {
    ASSERT_LT((size_t)h_index[i], key_vec.size());
    uint64_t from = key_vec[h_index[i]];
    Edge e;
    e.from = from;
    e.to = h_neighbors[i];
    ASSERT_NE(edge_map.find(e), edge_map.end());
    int edge_type = edge_map[e];
    int tmp_type = e.to / node_each;
    ASSERT_GE(edge_type, last_edge_type);
    last_edge_type = edge_type;
    ASSERT_LT(i, edges_split_num[edge_type]);
    if (edge_type != 0) {
      ASSERT_GE(i, edges_split_num[edge_type - 1]);
    }
    ASSERT_EQ(tmp_type, h_node_type[i]);
    ASSERT_EQ(query_edge_set.find(e), query_edge_set.end());
    query_edge_set.insert(e);
  }

  // hot_start
  for (int i = 0; i < 100; i++) {
    edges_split_num.clear();
    int64_t neighbor_len = 0;
    iter->SampleNeighbors(0,
                          (int64_t *)key,
                          key_vec.size(),
                          sample_size,
                          edges_split_num,
                          &neighbor_len,
                          edge_types.size(),
                          edge_type_graph_);
  }
  for (int i = 0; i < 100; i++) {
    edges_split_num.clear();
    edges_len = 0;
    iter->sample_neighbor_with_node_type(0,
                                         key,
                                         sample_size,
                                         key_vec.size(),
                                         edge_type_graph_,
                                         d_node_types,
                                         node_types.size(),
                                         edges_len,
                                         edges_split_num);
  }

  auto start2 = std::chrono::steady_clock::now();
  for (int i = 0; i < 100; i++) {
    edges_split_num.clear();
    int64_t neighbor_len = 0;
    iter->SampleNeighbors(0,
                          (int64_t *)key,
                          key_vec.size(),
                          sample_size,
                          edges_split_num,
                          &neighbor_len,
                          edge_types.size(),
                          edge_type_graph_);
  }
  auto end2 = std::chrono::steady_clock::now();
  auto tt =
      std::chrono::duration_cast<std::chrono::microseconds>(end2 - start2);
  std::cerr << "total time cost with all type query is " << tt.count() << " us"
            << std::endl;

  auto start1 = std::chrono::steady_clock::now();
  for (int i = 0; i < 100; i++) {
    edges_split_num.clear();
    edges_len = 0;
    iter->sample_neighbor_with_node_type(0,
                                         key,
                                         sample_size,
                                         key_vec.size(),
                                         edge_type_graph_,
                                         d_node_types,
                                         node_types.size(),
                                         edges_len,
                                         edges_split_num);
  }

  auto end1 = std::chrono::steady_clock::now();
  auto tt1 =
      std::chrono::duration_cast<std::chrono::microseconds>(end1 - start1);
  std::cerr << "total time cost with type info query is " << tt1.count()
            << " us" << std::endl;
}
