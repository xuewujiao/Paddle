#include "hip/hip_runtime.h"
// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/fleet/heter_ps/graph_bucket.h"
__global__ void add_new_keys(uint64_t *keys,
                             size_t set_size,
                             uint64_t *new_keys,
                             size_t n,
                             uint64_t unused_key) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    size_t index = new_keys[idx] % set_size;
    size_t counter = 0;
    while (counter < set_size) {
      uint64_t old = atomicCAS(keys + index, unused_key, new_keys[idx]);
      if (old == unused_key || old == new_keys[idx]) break;
      counter++;
      index++;
      if (index >= set_size) {
        index = 0;
      }
    }
  }
}
}

__global__ void initialize_keys(uint64_t *keys,
                                size_t set_size,
                                uint64_t unused_key) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < set_size) {
    keys[idx] = unused_key;
  }
}
}

__global__ void relocate_keys(uint64_t *keys,
                              size_t set_size,
                              uint64_t *new_keys,
                              size_t new_set_size,
                              uint64_t unused_key) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < set_size && keys[idx] != unused_key) {
    size_t index = keys[idx] % new_set_size;
    size_t counter = 0;
    while (counter < new_set_size) {
      uint64_t old = atomicCAS(new_keys + index, unused_key, keys[idx]);
      if (old == unused_key || old == keys[idx]) break;
      counter++;
      index++;
      if (index >= new_set_size) {
        index = 0;
      }
    }
  }
}

__global__ void cal_emb_dist(int n,
                             int *type,
                             float *emb,
                             int dim,
                             float *aggregated_emb,
                             float *output) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    int emb_type = type[idx];
    int aggregated_pos = emb_type * dim;
    int pos = dim * idx;
    float res = 0;
    for (int i = 0; i < dim; i++) {
      res += fabs(aggregated_emb[pos + i] - emb[pos + i]);
    }
    output[idx] = res;
  }
}

__global__ void export_sum(uint64_t *keys,
                           size_t set_size,
                           uint64_t unused_key,
                           size_t *sum,
                           size_t *block_offset) {
  __shared__ size_t local_num;
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();
  if (keys[idx] != unused_key) {
    atomicAdd(&local_num, 1);
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    block_offset[blockIdx.x] = atomicAdd(sum, local_num);
  }
}

void arrage_reduce_one_dim_data(int n, int *data, int *type) {
  thrust::device_ptr<int> data_ptr = thrust::device_pointer_cast(data);
  thrust::device_ptr<int> type_ptr = thrust::device_pointer_cast(type);
  thrust::sort_by_key(thrust::device, type_ptr, type_ptr + n, data_ptr);
}
void reduce_one_dim(
    int n, int type_len, int *data, int *type, int *res_data, int *res_type) {
  int len = n;
  int *dist_data[2], *dist_type[2];
  if (len == type_len) {
    vector<int> ind(type_len, 0);
    for (int i = 0; i < type_len; i++) ind[i] = i;
    sort(ind.begin(), ind.end(), [&](int a, int b) {
      return type[a] < type[b];
    });
    for (int i = 0; i < type_len; i++) {
      res_data[i] = data[ind[i]];
      res_type[i] = type[ind[i]];
    }
    return;
  }
  int cur = 0;
  hipMalloc((void **)&dist_data[0], sizeof(int) * (n + type_len));
  hipMalloc((void **)&dist_type[0], sizeof(int) * (n + type_len));
  hipMemcpy(dist_data[0], data, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dist_type[0], type, n * sizeof(int), hipMemcpyHostToDevice);
  hipMalloc((void **)&dist_data[1], sizeof(int) * (n + type_len));
  hipMalloc((void **)&dist_type[1], sizeof(int) * (n + type_len));
  arrage_reduce_one_dim_data(n, dist_data[0], dist_type[0]);
  int *raw_ptr;
  hipMalloc((void **)&raw_ptr, sizeof(int));
  hipStream_t stream;
  hipStreamCreate(&stream);
  int thread_num = 512;
  printf("start to test\n");
  while (len != type_len) {
    int block_num = (len - 1) / thread_num + 1;
    int p = block_num + type_len;
    int block_num2 = (p - 1) / thread_num + 1;
    fill_zero<<<block_num2, thread_num, 0, stream>>>(dist_data[1 - cur], p);
    hipMemcpyAsync(
        raw_ptr, &block_num, sizeof(int), hipMemcpyHostToDevice, stream);
    reduce_emb<<<block_num, thread_num, 0, stream>>>(dist_data[cur],
                                                     dist_type[cur],
                                                     len,
                                                     dist_data[1 - cur],
                                                     dist_type[1 - cur],
                                                     raw_ptr);
    hipMemcpyAsync(&len, raw_ptr, sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    cur = 1 - cur;
  }
  hipMemcpy(
      res_data, dist_data[cur], len * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(
      res_type, dist_type[cur], len * sizeof(int), hipMemcpyDeviceToHost);
}

__global__ void reduce_emb(
    int *data, int *type, int n, int *dist_data, int *dist_type, int *tot) {
  __shared__ int first_type;
  __shared__ int last_type;
  __shared__ int last_pos;
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    if (threadIdx.x == 0) {
      first_type = type[idx];
    } else if (idx == n - 1 || threadIdx.x == blockDim.x - 1) {
      last_type = type[idx];
      last_pos = idx;
    }
    int cur_type = type[idx];
    __syncthreads();
    size_t size = blockDim.x / 2;
    int temp;
    while (size != 0) {
      if (threadIdx.x >= size && cur_type == type[idx - size]) {
        temp = data[idx];
      }
      __syncthreads();
      if (threadIdx.x >= size && cur_type == type[idx - size]) {
        data[idx - size] += temp;
      }
      size /= 2;
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      dist_data[blockIdx.x] = data[idx];
      dist_type[blockIdx.x] = type[idx];
    }
    if (first_type != last_type) {
      if (threadIdx.x > 0 && type[idx] != type[idx - 1]) {
        if (type[idx] == last_type && last_pos < n - 1 &&
            type[last_pos + 1] == last_type) {
          int sum, old;
          do {
            sum = data[last_pos + 1];
            old = atomicCAS(data + last_pos + 1, sum, sum + data[idx]);
          } while (old != sum);
        } else {
          int t = atomicAdd(tot, 1);
          dist_data[t] = data[idx];
          dist_type[t] = type[idx];
        }
      }
    }
  }
}

__global__ void export_keys(uint64_t *keys,
                            size_t set_size,
                            uint64_t unused_key,
                            uint64_t *exported_keys,
                            size_t *block_offset) {
  __shared__ size_t local_num;
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadIdx.x == 0) {
    local_num = 0;
  }
  __syncthreads();
  if (keys[idx] != unused_key) {
    size_t pos = atomicAdd(&local_num, 1);
    export_keys[block_offset[blockIdx.x] + pos] = keys[idx];
  }
}

__global__ void find_keys(int n,
                          uint64_t *query_keys,
                          uint64_t *keys,
                          int *output) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    output[idx] = 0;
    size_t index = query_keys[idx] % set_size;
    size_t counter = 0;
    while (counter < set_size) {
      if (keys[index] == unused_key) break;
      if (keys[index] == query_keys[idx]) {
        output[idx] = 1;
        break;
      }
      counter++;
      index++;
      if (index >= set_size) {
        index = 0;
      }
    }
  }
}

__global__ void record_sum(int n,
                           int *output,
                           int *range,
                           int *neighbor_count) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < n) {
    if (idx == 0) {
      neighbor_count[idx] = output[range[idx] - 1];
    } else {
      neighbor_count[idx] = output[range[idx] - 1] - output[range[idx - 1] - 1];
    }
  }
}

void GraphBucket::init_keys() {
  keys_alloc_ = memory::AllocShared(place_, capacity * sizeof(uint64_t));
  keys_ = reinterpret_cast<int64_t *>(keys_alloc_->ptr());
  initialize_keys<<<GET_BLOCKS(capacity), CUDA_NUM_THREADS>>>(
      keys, capacity, unused_key);
  emb_alloc_ =
      memory::AllocShared(place_, type_size_ * emb_size_ * sizeof(float));
  aggregated_emb = reinterpret_cast<float *>(emb_alloc_->ptr());
  thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(aggregated_emb);
  const auto &exec_policy = thrust::cuda::par(allocator).on(stream_);
  thrust::fill(exec_policy, dev_ptr, dev_ptr + type_size_ * emb_size_, (int)0);
}

void GraphBucket::get_neighbor_count(int unique_count,
                                     int n,
                                     uint64_t *neighbors,
                                     int *range,
                                     int *neighbor_count) {
  auto res = memory::AllocShared(place_, n * sizeof(int));
  int *output = reinterpret_cast<int64_t *>(res->ptr());
  find_keys<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, neighbors, keys, output);
  thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(output);
  const auto &exec_policy = thrust::cuda::par(allocator).on(stream_);
  thrust::inclusive_scan(exec_policy, dev_ptr, dev_ptr + n, dev_ptr);
  record_sum<<<GET_BLOCKS(unique_count), CUDA_NUM_THREADS>>>(
      unique_count, output, range, neighbor_count);
}
